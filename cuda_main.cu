#include "hip/hip_runtime.h"

#include "cuda_main.cuh"

//a*rows+x
void matrix_mul_cpu(int i, int j, float* M, int* N, float* P, int width)
{

    //if (j==0) printf("SP:%d %d\n", i, j);
    int k;
    int x;
    int front = -1, rear = -1, size = 0;
    f_patch* Patches = (f_patch*)malloc(sizeof(f_patch) * width);
    int* Q = (int*)malloc(sizeof(int) * width);
    for (k = 0; k < width; k++) {
        Patches[k].weight = M[i * width + k];
        Patches[k].IsDiscover = 0;
        Q[k] = INF;
    }
    Q[0] = i;
    front = 0;
    rear = 0;
    size = 1;
    while (size != 0) {
        x = Q[front]; front = (front + 1) % width; size--;
        if (Patches[x].IsDiscover) continue;
        if (x == j) { P[i * width + j] = Patches[j].weight; return; }
        Patches[x].IsDiscover = 1;
        for (int a = 0; a < 3; a++) {
            if (Patches[N[a * 4000 + x]].weight == -10) { Patches[N[a * 4000 + x]].weight = Patches[x].weight + M[x * width + N[a * 4000 + x]]; }
            else Patches[N[a * 4000 + x]].weight = Patches[N[a * 4000 + x]].weight <= Patches[x].weight + M[x * width + N[a * 4000 + x]]
                ? Patches[N[a * 4000 + x]].weight : Patches[x].weight + M[x * width + N[a * 4000 + x]];
        }
        int max = -1;
        for (int a = 0; a < width; a++) {
            if (!Patches[a].IsDiscover) {
                if (max == -1) max = a;
                else if (Patches[a].weight!=-10) max = Patches[max].weight <= Patches[a].weight ? max : a;
            }
        }
        rear = (rear + 1) % width; Q[rear] = max; size++;
    }
}

__global__ void matrix_mul_gpu(float* M, int* N, float* P, int width)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    //if (j==0) printf("SP:%d %d\n", i, j);
    int k;
    int x;
    int front = -1, rear = -1, size = 0;
    f_patch* Patches = (f_patch*)malloc(sizeof(f_patch) * width);
    int* Q = (int*)malloc(sizeof(int) * width);
    for (k = 0; k < width; k++) {
        Patches[k].weight = M[i * width + k];
        Q[k] = INF;
    }
    Q[0] = i;
    front = 0;
    rear = 0;
    size = 1;
    while (size != 0) {
        x = Q[front]; front = (front + 1) % width; size--;
        if (Patches[x].IsDiscover) continue;
        if (x == j) { P[i * width + j] = Patches[j].weight; return; }
        Patches[x].IsDiscover = 1;
        for (int a = 0; a < 3; a++) {
            if (Patches[N[x * 3 + a]].weight == -10) { Patches[N[x * 3 + a]].weight = Patches[x].weight + M[x * width + N[x * 3 + a]]; }
            else Patches[N[x * 3 + a]].weight = Patches[N[x * 3 + a]].weight <= Patches[x].weight + M[x * width + N[x * 3 + a]]
                ? Patches[N[x * 3 + a]].weight : Patches[x].weight + M[x * width + N[x * 3 + a]];
        }
        int max = -1;
        for (int a = 0; a < width; a++) {
            if (!Patches[a].IsDiscover) {
                if (max == -1) max = a;
                else if (Patches[a].weight != -10) max = Patches[max].weight <= Patches[a].weight ? max : a;
            }
        }
        rear = (rear + 1) % width; Q[rear] = max; size++;
    }
}


void GPUForDis(int Row, int Col, float* weights , int* Dual, float* Dist)
{
    //struct timeval start, end;
    //gettimeofday(&start, NULL);
    /*
    float* weights = (float*)malloc(sizeof(float) * Row * Col);
    int* Dual = (int*)malloc(sizeof(int) * Row * 3);
    float* Dist = (float*)malloc(sizeof(float) * Row * Col);*/
    //malloc device memory
    float* d_dataWei, * d_dataDist;
    int* d_dataDual;
    /*hipMalloc((void**)&d_dataWei, sizeof(float) * Row * Col);
    hipMalloc((void**)&d_dataDual, sizeof(int) * Row * 3);
    hipMalloc((void**)&d_dataDist, sizeof(float) * Row * Col);
    //set value
    

    hipMemcpy(d_dataWei, weights, sizeof(float) * Row * Col, hipMemcpyHostToDevice);
    hipMemcpy(d_dataDual, Dual, sizeof(int) * Row * 3, hipMemcpyHostToDevice);
    //dim3 threadPerBlock(16, 16);
    dim3 threadPerBlock(16, 16);
    //dim3 blockNumber(2,2);
    dim3 blockNumber((Col + threadPerBlock.x - 1) / threadPerBlock.x, (Row + threadPerBlock.y - 1) / threadPerBlock.y);
    printf("Block(%d,%d)   Grid(%d,%d).\n", threadPerBlock.x, threadPerBlock.y, blockNumber.x, blockNumber.y);
    matrix_mul_gpu << <blockNumber, threadPerBlock >> > (d_dataWei, d_dataDual, d_dataDist, Col);
    //������������-һ������ָ��
    hipMemcpy(Dist, d_dataDist, sizeof(float) * Row * Col, hipMemcpyDeviceToHost);*/
    for (int i = 0; i < 4000; i++) {
        for (int j = 0; j < 4000; j++) {
            matrix_mul_cpu(i, j, weights, Dual,Dist, 4000);
        }
    }
    for (int i = 0; i < 4; i++) {
        printf("%f %f %f %f\n", Dist[i * 4 + 0], Dist[i * 4 + 1], Dist[i * 4 + 2], Dist[i * 4 + 3]);
    }
    //�ͷ��ڴ�
    
    hipFree(d_dataWei);
    hipFree(d_dataDual);
    hipFree(d_dataDist);

    /*gettimeofday(&end, NULL);
    int timeuse = 1000000 * (end.tv_sec - start.tv_sec) + end.tv_usec - start.tv_usec;
    printf("total time is %d ms\n", timeuse / 1000);*/

}